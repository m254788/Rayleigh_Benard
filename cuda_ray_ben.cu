
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <math.h>

void circshift(int* out, int* in, int numrows, int numcols, int rowshift, int colshift) {
	for (int i = 0; i < numrows; i++){
		int ii = (i + rowshift)%numrows;
		if (ii<0) ii=numrows+ii;
		for (int j = 0; j < numcols; j++){
			int jj = (j + colshift) % numcols;
			if (jj<0) jj = numcols + jj;
			out[ii*numcols + jj] = in[i*numcols + j];
		}	
	}
}
/*
__device__ void columnSum(float* sum, float* mat, int rows, int cols){
	for(int j = 0; j < cols; j++){
		float colsum = 0;
		for(int i = 0; i < rows; i++){
			colsum += mat[i*cols+j];
		}
		sum[j] = colsum;
	}
}
*/

void printMat(float* mat, int rows, int cols){
        for(int i = 0; i < rows; i++){
                for(int j = 0; j < cols; j++){
                        std::cout << mat[i*cols+j]<<" ";
                }
                std::cout<<"\n";
        }
	std::cout<<"\n";
}
void write_vector_to_file(float* vector, int dim, int evolution) {
	std::ofstream myfile;
	myfile.open("evolution"+std::to_string(evolution)+".txt");
	for(int i = 0; i < dim; i++){
		myfile << vector[i] << "\n";
	}
}

__global__ void timestep(float* fIn, float* fOut,float* fEq,float* force, float* rho, float* T, float* ux, float* uy, float* tIn, float* tOut, float *tEq, int lx, int ly, int* cxNS, int* cyNS,int* cxT, int* cyT, float* tNS,float* tT, float Thot, int Tcold, float omegaNS, float omegaT, int* oppNS, int* stmNS, int* stmT){
	int nid = threadIdx.x + blockIdx.x*blockDim.x;
	int nnodes = lx*ly;
	if (nid < nnodes){
		bool bottom_node = (nid<lx);
		bool top_node = (nid>=(nnodes-lx));

		rho[nid] = 0;
		for(int i = 0; i < 9; i++){
			rho[nid] += fIn[i*nnodes + nid];
		}
		T[nid] = 0;
		for(int i = 0; i < 5; i++){
			T[nid] += tIn[i*nnodes + nid];
		}

		ux[nid] = (fIn[1*nnodes+nid]-fIn[3*nnodes+nid]+fIn[5*nnodes+nid]-fIn[6*nnodes+nid]-fIn[7*nnodes+nid]+fIn[8*nnodes+nid])/rho[nid];
		uy[nid] = (fIn[2*nnodes+nid]-fIn[4*nnodes+nid]+fIn[5*nnodes+nid]+fIn[6*nnodes+nid]-fIn[7*nnodes+nid]-fIn[8*nnodes+nid])/rho[nid];
		//collision
		for(int spd = 0; spd < 9; spd++){
			float cu = 3*(cxNS[spd]*ux[nid] + cyNS[spd]*uy[nid]);
			fEq[spd*nnodes + nid] = tNS[spd]*rho[nid]*(1+cu+(0.5)*cu*cu - (1.5)*(ux[nid]*ux[nid]+uy[nid]*uy[nid]));
			force[spd*nnodes + nid] = 3*tNS[spd]*rho[nid]*(T[nid]-((Thot+Tcold)/2))*(cyNS[spd]*0.001)/(Thot-Tcold);
			fOut[spd*nnodes + nid] = fIn[spd*nnodes+nid]-omegaNS*(fIn[spd*nnodes+nid]-fEq[spd*nnodes+nid])+force[spd*nnodes+nid];
		}
		
		for(int i = 0; i < 5; i++){
			float cu = 3*(cxT[i]*ux[nid] + cyT[i]*uy[nid]);
			tEq[i*nnodes+nid] = T[nid]*tT[i]*(1+cu);
			tOut[i*nnodes+nid] = tIn[i*nnodes+nid]-omegaT*(tIn[i*nnodes+nid]-tEq[i*nnodes+nid]);
		}

		//micro boundary fluid
		if (bottom_node || top_node){
			for(int i = 0; i < 9; i++){
				fOut[i*nnodes+nid] = fIn[oppNS[i]*nnodes + nid];
			}
		}

		//streaming
		for(int i = 0; i < 9; i++){
			fIn[i*nnodes+stmNS[i*nnodes+nid]] = fOut[i*nnodes+nid];
		}
		for(int i = 0; i < 5; i++){
			tIn[i*nnodes+(stmNS[i*nnodes+nid])] = tOut[i*nnodes+nid];
		}

		//micro boundary temp
		if(top_node){
			tIn[4*nnodes+nid] = Tcold-tIn[0*nnodes+nid]-tIn[1*nnodes+nid]-tIn[2*nnodes+nid]-tIn[3*nnodes+nid];
		}
		if(bottom_node){
			tIn[2*nnodes+nid] = Thot-tIn[0*nnodes+nid]-tIn[1*nnodes+nid]-tIn[3*nnodes+nid]-tIn[4*nnodes+nid];
		}
	
	}

}


int main(int argc, char* argv[]) {
	//give to device as arguments in kernel
	int ly = 51;//kernel arg
	int aspect_ratio = 2;
	int lx = ly*aspect_ratio; //kernel arg
	int nnodes = lx*ly; //kernel arg
	float delta_x = 1.0/(ly-2);
	float Pr = 1.0;
	float Ra = 100000;
	float gr = 0.001;
	float Thot = 1.0; //kernel arg
	int Tcold = 0; //kernel arg
	//float T0 = (Thot+Tcold)/2;
	float delta_t = sqrt(gr*delta_x);
	float nu = (sqrt(Pr/Ra)*delta_t)/(delta_x*delta_x);
	float k = sqrt(1.0/(Pr*Ra))*delta_t/(delta_x*delta_x);
	float omegaNS = 1.0/(3*nu + 0.5);//kernel arg
	float omegaT = 1.0/(3*k + 0.5);//kernel arg
	
	//host variables
	int maxT = 10000;
	//int Vis_ts = 100;
	//int Vis_ind = 0;
	
	//device needs to know these values
	float tNS[] = {4./9,1./9,1./9,1./9,1./9,1./36,1./36,1./36,1./36};//kernel arg
	int cxNS[] = {0,1,0,-1,0,1,-1,-1,1};//kernel arg
	int cyNS[] = {0,0,1,0,-1,1,1,-1,-1};//kernel arg
	int oppNS[] = {0,3,4,1,2,7,8,5,6};//kernel arg

	
	float *tNS_d;
	int *cxNS_d, *cyNS_d, *oppNS_d;
	hipMalloc((void**)&tNS_d,9*sizeof(float));
	hipMalloc((void**)&cxNS_d, 9*sizeof(int));
	hipMalloc((void**)&cyNS_d, 9*sizeof(int));
	hipMalloc((void**)&oppNS_d, 9*sizeof(int));
	hipMemcpy(tNS_d, tNS, 9*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(cxNS_d, cxNS, 9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(cyNS_d, cyNS, 9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(oppNS_d, oppNS, 9*sizeof(int),hipMemcpyHostToDevice);
	

	float tT[] = {1./3,1./6,1./6,1./6,1./6};//kernel arg
	int cxT[] = {0,1,0,-1,0};//kernel arg
	int cyT[] = {0,0,1,0,-1};//kernel arg
	//int oppT[] = {0,3,4,1,2};//kernel arg
	
	float *tT_d;
	int *cxT_d, *cyT_d;
	hipMalloc(&tT_d,5*sizeof(float));
	hipMalloc(&cxT_d,5*sizeof(int));
	hipMalloc(&cyT_d,5*sizeof(int));
	
	hipMemcpy(tT_d, tT, 5*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(cxT_d, cxT, 5*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(cyT_d, cyT, 5*sizeof(int),hipMemcpyHostToDevice);
	
	
	
	
	
	/*identify top and bottom nodes
	int* top_nodes = new int [lx];
	int* bottom_nodes = new int [lx];
	for(int i = 0; i < lx; i++){
		bottom_nodes[i] = i;
		top_nodes[i] = nnodes-lx+i;
	}
	
	int *top_nodes_d, *bottom_nodes_d;
	cudaMalloc((void**)&top_nodes_d,lx*sizeof(int));
	cudaMalloc((void**)&bottom_nodes_d,lx*sizeof(int));
	cudaMemcpy(top_nodes_d,top_nodes,lx*sizeof(int),cudaMemcpyHostToDevice);
	cudaMemcpy(bottom_nodes_d,bottom_nodes,lx*sizeof(int),cudaMemcpyHostToDevice);
	*/

	//initialize stuff
	float* fEven = new float [9*nnodes]; //initialize fIn
	for(int s = 0; s < 9; s++){
		for(int n = 0; n < nnodes; n++){
			fEven[s*nnodes+n] = tNS[s];
		}
	}

	float* tEven = new float [5*nnodes]; //initialize tIn
	for(int s = 0; s < 5; s++){
		for(int n = 0; n < nnodes; n++){
			tEven[s*nnodes+n] = tT[s]*Tcold;
		}
	}

	for(int s = 0; s < 5; s++){ //go along bottom nodes, set equal to Thot
		for(int b = 0; b < lx; b++){ 
			tEven[s*nnodes+b] = tT[s]*Thot;
		}
	}
	
	for(int s = 0; s < 5; s++){ //create asymmetry
		tEven[s*nnodes+int(1.5*lx)] = tT[s]*(1.1*Thot);
	}
	
	
	//create stream target matrices
	
	int* stmNS = new int [9*nnodes];
	int* stmT = new int [5*nnodes];
	int* ind = new int [nnodes];
	int* tInd = new int [nnodes];
	for (int i = 0; i < nnodes; i++){
		ind[i] = i;
	}
	for (int i = 0; i < 9; i++){
		circshift(tInd, ind, ly, lx, -cyNS[i], -cxNS[i]);
		for(int n = 0; n < nnodes; n++){
			stmNS[i*nnodes+n] = tInd[n];
		}
		if (i < 5){
			circshift(tInd, ind, ly, lx, -cyT[i], -cxT[i]);
			for(int n = 0; n < nnodes; n++){
				stmT[i*nnodes+n] = tInd[n];
			}
		}
	}
	
	delete[] ind;
	delete[] tInd;
	
	
	int *stmNS_d, *stmT_d;
	hipMalloc(&stmNS_d, 9*nnodes*sizeof(int));
	hipMalloc(&stmT_d, 9*nnodes*sizeof(int));
	hipMemcpy(stmNS_d, stmNS, 9*nnodes*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(stmT_d, stmT, 5*nnodes*sizeof(int), hipMemcpyHostToDevice);
	
	
	float* T = new float[nnodes];
	float* ux = new float[nnodes];
	float* uy = new float[nnodes];
	//here down not actually needed just for debugging purposes
	float* rho = new float[nnodes];
	float* fEq = new float[9*nnodes];
	float* fOut = new float[9*nnodes];
	float* force = new float[9*nnodes];
	float* tEq = new float[5*nnodes];
	float* tOut = new float[5*nnodes];
	//here up ''

	float *rho_d, *T_d, *ux_d, *uy_d, *fEven_d, *fOdd_d, *fEq_d, *force_d, *tEven_d, *tOdd_d, *tEq_d;
	
	hipMalloc(&rho_d, nnodes*sizeof(float));
	hipMalloc(&T_d, nnodes*sizeof(float));
	hipMalloc(&ux_d, nnodes*sizeof(float));
	hipMalloc(&uy_d, nnodes*sizeof(float));
	hipMalloc(&fEven_d, 9*nnodes*sizeof(float));
	hipMalloc(&fOdd_d, 9*nnodes*sizeof(float));
	hipMalloc(&fEq_d, 9*nnodes*sizeof(float));
	hipMalloc(&force_d, 9*nnodes*sizeof(float));
	hipMalloc(&tEven_d, 5*nnodes*sizeof(float));
	hipMalloc(&tOdd_d, 5*nnodes*sizeof(float));
	hipMalloc(&tEq_d, 5*nnodes*sizeof(float));

	
	hipMemcpy(fEven_d, fEven, 9*nnodes*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(tEven_d, tEven, 5*nnodes*sizeof(float), hipMemcpyHostToDevice);
	
	for(int cycle = 0; cycle < maxT; cycle++){
		timestep<<<((nnodes+127)/128),128>>>(fEven_d,fOdd_d,fEq_d,force_d, rho_d, T_d, ux_d, uy_d, tEven_d, tOdd_d, tEq_d, lx, ly, cxNS_d, cyNS_d,cxT_d,cyT_d, tNS_d,tT_d, Thot, Tcold, omegaNS, omegaT, oppNS_d, stmNS_d,stmT_d);

	}
	//just for debugging
	hipMemcpy(T,T_d,nnodes*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(ux,ux_d,nnodes*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(uy,uy_d,nnodes*sizeof(float),hipMemcpyDeviceToHost);
	/*
	cudaMemcpy(rho,rho_d,nnodes*sizeof(float),cudaMemcpyDeviceToHost);
	cudaMemcpy(fEq,fEq_d,9*nnodes*sizeof(float),cudaMemcpyDeviceToHost);
	cudaMemcpy(fOut,fOdd_d,9*nnodes*sizeof(float),cudaMemcpyDeviceToHost);
	cudaMemcpy(force,force_d,9*nnodes*sizeof(float),cudaMemcpyDeviceToHost);
	cudaMemcpy(tEq,tEq_d,5*nnodes*sizeof(float),cudaMemcpyDeviceToHost);
	cudaMemcpy(tOut,tOdd_d,5*nnodes*sizeof(float),cudaMemcpyDeviceToHost);
	*/
	
	printMat(T,1,nnodes);
	printMat(ux,1,nnodes);
	printMat(uy,1,nnodes);



	hipFree(rho_d);
	hipFree(T_d);
	hipFree(ux_d);
	hipFree(uy_d);
	hipFree(fEven_d);
	hipFree(fOdd_d);
	hipFree(fEq_d);
	hipFree(force_d);
	hipFree(tEven_d);
	hipFree(tOdd_d);
	hipFree(tEq_d);

	delete[] fEven;
	delete[] tEven;
	delete[] stmNS;
	delete[] stmT;	
	delete[] T;
	delete[] ux;
	delete[] uy;
	delete[] rho;
	delete[] fEq;
	delete[] fOut;
	delete[] force;
	delete[] tEq;
	delete[] tOut;

}
